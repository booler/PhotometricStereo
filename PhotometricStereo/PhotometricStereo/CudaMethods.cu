#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void GetNearestPixelIndexKernel_Manhaton(double *ball_pixel, double *object, double *shadow, 
                                           double *weight, int *result, int ball_pixel_count, int dimension,
                                           int dark_threoshold, int shadow_threoshold, int start, int end)
{
    int index = start + blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= end)
        return;
    object += (index * dimension);
    shadow += (index * dimension);

    bool dark = true;
    for (int i = 0; i < dimension; ++i)
        if (object[i] > dark_threoshold)
        {
            dark = false;
            break;
        }
    if (dark)
    {
        result[index] = -1;
        return;
    }

    double minimun_dis = 1.0e300;
    double minimum_idx = -1;
    for (int i = 0; i < ball_pixel_count; ++i)
    {
        double dis = 0;
        bool shadow_flag = true;
        for (int j = 0; j < dimension; ++j)
        {
            if ((shadow[j] > 0 ? shadow[j] : -shadow[j]) > shadow_threoshold)
                continue;
            shadow_flag = false;
            dis += ((object[j] > ball_pixel[j]) ? (object[j] - ball_pixel[j]) : (ball_pixel[j] - object[j])) * weight[static_cast<int>(object[j])];
        }
        if (!shadow_flag)
        {
            if (minimun_dis > dis)
            {
                minimun_dis = dis;
                minimum_idx = i;
            }
        }
        ball_pixel += dimension;
    }
    result[index] = minimum_idx;
    return;
}

__global__ void GetNearestPixelIndexKernel_Angle(double *ball_pixel, double *object, double *shadow, 
                                           int *result, int ball_pixel_count, int dimension,
                                           int dark_threoshold, int shadow_threoshold, int start, int end)
{
    int index = start + blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= end)
        return;
    object += (index * dimension);
    shadow += (index * dimension);

    bool dark = true;
    for (int i = 0; i < dimension; ++i)
        if (object[i] > dark_threoshold)
        {
            dark = false;
            break;
        }
    if (dark)
    {
        result[index] = -1;
        return;
    }

    double maximum_dis = -1.0e300;
    double maximum_idx = -1;
    for (int i = 0; i < ball_pixel_count; ++i)
    {
        double dis = 0;
        double l1 = 0;
        bool shadow_flag = true;
        for (int j = 0; j < dimension; ++j)
        {
            if ((shadow[j] > 0 ? shadow[j] : -shadow[j]) > shadow_threoshold)
                continue;
            shadow_flag = false;
            dis += object[j] * ball_pixel[j];
            l1 += ball_pixel[j] * ball_pixel[j];
        }
        if (!shadow_flag)
        {
            dis = dis / sqrt(l1);
            if (maximum_dis < dis)
            {
                maximum_dis = dis;
                maximum_idx = i;
            }
        }
        ball_pixel += dimension;
    }
    result[index] = maximum_idx;
    return;
}

__global__ void GetNearestPixelIndexKernel_Manhaton_NoShadow(double *ball_pixel, double *object, 
                                           double *weight, int *result, int ball_pixel_count, int dimension,
                                           int dark_threoshold, int start, int end)
{
    int index = start + blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= end)
        return;
    object += (index * dimension);

    bool dark = true;
    for (int i = 0; i < dimension; ++i)
        if (object[i] > dark_threoshold)
        {
            dark = false;
            break;
        }
    if (dark)
    {
        result[index] = -1;
        return;
    }

    double minimun_dis = 1.0e300;
    double minimum_idx = -1;
    for (int i = 0; i < ball_pixel_count; ++i)
    {
        double dis = 0;
        for (int j = 0; j < dimension; ++j)
            dis += ((object[j] > ball_pixel[j]) ? (object[j] - ball_pixel[j]) : (ball_pixel[j] - object[j])) * weight[static_cast<int>(object[j])];
        if (minimun_dis > dis)
        {
            minimun_dis = dis;
            minimum_idx = i;
        }
        ball_pixel += dimension;
    }
    result[index] = minimum_idx;
    return;
}

__global__ void GetNearestPixelIndexKernel_Angle_NoShadow(double *ball_pixel, double *object, 
                                           int *result, int ball_pixel_count, int dimension,
                                           int dark_threoshold, int start, int end)
{
    int index = start + blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= end)
        return;
    object += (index * dimension);

    bool dark = true;
    for (int i = 0; i < dimension; ++i)
        if (object[i] > dark_threoshold)
        {
            dark = false;
            break;
        }
    if (dark)
    {
        result[index] = -1;
        return;
    }

    double maximum_dis = -1.0e300;
    double maximum_idx = -1;
    for (int i = 0; i < ball_pixel_count; ++i)
    {
        double dis = 0;
        double l1 = 0;
        for (int j = 0; j < dimension; ++j)
        {
            dis += object[j] * ball_pixel[j];
            l1 += ball_pixel[j] * ball_pixel[j];
        }
        dis = dis / sqrt(l1);
        if (maximum_dis < dis)
        {
            maximum_dis = dis;
            maximum_idx = i;
        }
        ball_pixel += dimension;
    }
    result[index] = maximum_idx;
    return;
}

hipError_t CudaGetNearestPixelIndex(double *ball_pixel, double *object, double *shadow, 
                                 double *weight, int *result, int ball_pixel_count, int h, int w, int dimension,
                                 int dark_threoshold, int shadow_threoshold, bool manhaton)
{
    typedef double LightningType;
    hipError_t cuda_status;
    int object_size = w * h;
    int ball_pixel_lighning_vector_size = dimension * ball_pixel_count;
    int object_pixel_lightning_vector_size = dimension * object_size;
    int object_pixel_shadow_vector_size = object_pixel_lightning_vector_size;
    const int gray_scale_weight_size = 256;

    LightningType *ptr_ball_pixel = nullptr, 
           *ptr_object_pixel = nullptr, 
           *ptr_object_shadow = nullptr;

    int *ptr_result = nullptr;
    double *ptr_gray_scale_weight = nullptr;

    int per_batch_count = 399999999 / ball_pixel_count / dimension + 1;
    int batch_count = ((object_size % per_batch_count == 0) ? 0 : 1) + object_size / per_batch_count;

    int threads_num = 256;
    int blocks_num = per_batch_count / threads_num + (per_batch_count % threads_num == 0 ? 0 : 1);

    int i;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess) {
        goto Error;
    }

    cuda_status = hipMalloc((void**)&ptr_ball_pixel, ball_pixel_lighning_vector_size * sizeof(LightningType));
    if (cuda_status != hipSuccess)
        goto Error;
    cuda_status = hipMemcpy(ptr_ball_pixel, ball_pixel, ball_pixel_lighning_vector_size * sizeof(LightningType), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess)
        goto Error;   

    cuda_status = hipMalloc((void**)&ptr_object_pixel, object_pixel_lightning_vector_size * sizeof(LightningType));
    if (cuda_status != hipSuccess)
        goto Error;
    cuda_status = hipMemcpy(ptr_object_pixel, object, object_pixel_lightning_vector_size * sizeof(LightningType), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess)
        goto Error;   

    if (shadow != nullptr)
    {
        cuda_status = hipMalloc((void**)&ptr_object_shadow, object_pixel_shadow_vector_size * sizeof(LightningType));
        if (cuda_status != hipSuccess)
            goto Error;
        cuda_status = hipMemcpy(ptr_object_shadow, shadow, object_pixel_shadow_vector_size * sizeof(LightningType), hipMemcpyHostToDevice);
        if (cuda_status != hipSuccess)
            goto Error;   
    }

    if (weight != nullptr)
    {
        cuda_status = hipMalloc((void**)&ptr_gray_scale_weight, gray_scale_weight_size * sizeof(double));
        if (cuda_status != hipSuccess)
            goto Error;
        cuda_status = hipMemcpy(ptr_gray_scale_weight, weight, gray_scale_weight_size * sizeof(double), hipMemcpyHostToDevice);
        if (cuda_status != hipSuccess)
            goto Error;   
    }

    cuda_status = hipMalloc((void**)&ptr_result, object_size * sizeof(int));
    if (cuda_status != hipSuccess)
        goto Error;

    /*cuda_status = cudaMemcpy(ptr_result, result, object_size * sizeof(int), cudaMemcpyHostToDevice);
    if (cuda_status != cudaSuccess)
        goto Error;   */

    for (i = 0; i < batch_count; ++i)
    {
        int start = i * per_batch_count;
        int end = (i + 1) * per_batch_count > object_size ? object_size : (i + 1) * per_batch_count;
        if (manhaton && shadow != nullptr)
            GetNearestPixelIndexKernel_Manhaton<<<blocks_num, threads_num>>>(ptr_ball_pixel,
                                                        ptr_object_pixel,
                                                        ptr_object_shadow,
                                                        ptr_gray_scale_weight,
                                                        ptr_result, 
                                                        ball_pixel_count, 
                                                        dimension,
                                                        dark_threoshold,
                                                        shadow_threoshold,
                                                        start,
                                                        end                                                        
                                                        );
        else if (!manhaton && shadow != nullptr)
            GetNearestPixelIndexKernel_Angle<<<blocks_num, threads_num>>>(ptr_ball_pixel,
                                                        ptr_object_pixel,
                                                        ptr_object_shadow,
                                                        ptr_result, 
                                                        ball_pixel_count, 
                                                        dimension,
                                                        dark_threoshold,
                                                        shadow_threoshold,
                                                        start,
                                                        end   
                                                        );
        else if (manhaton && shadow == nullptr)
            GetNearestPixelIndexKernel_Manhaton_NoShadow<<<blocks_num, threads_num>>>(ptr_ball_pixel, 
                                                         ptr_object_pixel,
                                                         ptr_gray_scale_weight, 
                                                         ptr_result,
                                                         ball_pixel_count,
                                                         dimension, 
                                                         dark_threoshold, 
                                                         start,
                                                         end
                                                         );
        else if (!manhaton && shadow == nullptr)
            GetNearestPixelIndexKernel_Angle_NoShadow<<<blocks_num, threads_num>>>(ptr_ball_pixel,
                                                      ptr_object_pixel,
                                                      ptr_result,
                                                      ball_pixel_count,
                                                      dimension, 
                                                      dark_threoshold,
                                                      start,
                                                      end
                                                      );


                                                         
              // Check for any errors launching the kernel
        cuda_status = hipGetLastError();
        if (cuda_status != hipSuccess) 
            goto Error;
    
        // cudaDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cuda_status = hipDeviceSynchronize();
        if (cuda_status != hipSuccess) 
            goto Error;

      }

      // Copy output vector from GPU buffer to host memory.
    cuda_status = hipMemcpy(result, ptr_result, object_size * sizeof(int), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) 
        goto Error;

    return cuda_status;
Error:
    hipFree(ptr_ball_pixel);
    hipFree(ptr_object_shadow);
    hipFree(ptr_object_pixel);
    hipFree(ptr_gray_scale_weight);
    hipFree(ptr_result);
    return cuda_status;
}


  __global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    c[i] = a[i] + b[i];
}

 // Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    static const int threadNum = 256;
    
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;
    int blockNum = size / threadNum + ((size % threadNum == 0) ? 0 : 1);
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

      // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<blockNum, threadNum>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
